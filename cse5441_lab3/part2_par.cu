
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#define dim 4096

__global__ void kernel(double **A, double** C) {
    int idx = threadIdx.x * 4;
    int jdx = blockIdx.x * 4;
    for (int k = 0; k < dim; k++) {
        for (int i = idx; i < idx + 4; i++)
            for (int j = jdx; j < jdx + 2; j++)
                C[i][j] += A[k][i] * A[k][j];
    }
}

int main() {
    double **A;    // host pointer operand
    double **C;    // host pointer result
    double **d_a;  // device pointer operand
    double **d_c;  // device pointer result

    // thread hierarchy
    int nblocks = 1024;
    int tpb = 1024;

    // allocate memory
    size_t memSize;
    A = new double*[dim];
    C = new double*[dim];
    memSize = dim * dim * sizeof(double);
    hipMalloc((void***)&d_a, memSize);
    hipMalloc((void***)&d_c, memSize);

    // generate random array & copy
    for (int i = 0; i < dim; i++) {
        A[i] = new double[dim];
        C[i] = new double[dim];
        for (int j = 0; j < dim; j++) {
            C[i][j] = 0.0;
            A[i][j] = 1.0 + ((double)rand() / (double)RAND_MAX);
        }
    }
    hipMemcpy(d_a,A,memSize,hipMemcpyHostToDevice);
    hipMemcpy(d_c,C,memSize,hipMemcpyHostToDevice);

    // launch kernel
    dim3 dimGrid(tpb);
    dim3 dimBlock(nblocks);
    kernel<<<dimGrid,dimBlock>>>(d_a,d_c);

    hipError_t err = hipGetLastError();
    if (err != hipSuccess) 
        printf("Error: %s\n", hipGetErrorString(err));
    else
        printf("Success: terminating!\n");
}

