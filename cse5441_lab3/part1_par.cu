
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#define dim 4097

void kernel(double **F) {
    for (int k = 0; k < 100; k++)
        for (int i = 1; i < dim; i++)
            for (int j = 0; j < dim - 1; j++)
                F[i][j] = F[i-1][j+1] + F[i][j+1];
}

int main() {
    double **F;    // host pointer
    double **d_a;  // device pointer

    // thread hierarchy
    int nblocks = 1024;
    int tpb = 1024;

    // allocate memory
    size_t memSize;
    F = new double*[dim];
    memSize = dim * dim * sizeof(double);
    hipMalloc((void***)&d_a, memSize);

    // generate random array
    for (int i = 0; i < dim; i++) {
        F[i] = new double[dim];
        for (int j = 0; j < dim; j++) {
            F[i][j] = 1.0 + ((double)rand() / (double)RAND_MAX);
        }
    }
    hipMemcpy(d_a,F,memSize,hipMemcpyHostToDevice);
    // call kernel
    kernel(F);
}

